#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "pcisph.h"
#include "robin_hood.h"
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstring>
#include <iostream>
#include <thrust/extrema.h>
#include <vector>

#define get_tid()                                                              \
  (blockDim.x * (blockIdx.x + blockIdx.y * gridDim.x) + threadIdx.x)

PCISPH::PCISPH(const Float xmin, const Float xmax, const Float ymin,
               const Float ymax, const Float zmin, const Float zmax,
               const size_t grid_size_x, const size_t grid_size_y,
               const size_t grid_size_z)
    : xmin(xmin), xmax(xmax), ymin(ymin), ymax(ymax), zmin(zmin), zmax(zmax),
      grid_size_x(grid_size_x), grid_size_y(grid_size_y),
      grid_size_z(grid_size_z) {}

int bs = 256;
dim3 grid_every_particle;
dim3 grid_every_component;

void PCISPH::init(int n, Float delta) {
  auto initRes = hipInit(0);
  if (initRes != hipSuccess) {
    std::cerr << "CUDA init failed" << std::endl;
    exit(1);
  }
  std::cerr << "CUDA init success" << std::endl;
  std::cerr << "For each particle: ";
  std::cerr << "bs = " << bs << ' ';
  grid_every_particle = (n + bs - 1) / bs;
  grid_every_component = (n * 3 + bs - 1) / bs;
  std::cerr << "grid = (" << grid_every_particle.x << ','
            << grid_every_particle.y << ',' << grid_every_particle.z << ')'
            << std::endl;
  std::cerr << "number of particles = " << n << std::endl;

  this->n = n;
  this->delta = delta;
  hipMallocManaged(&x, sizeof(Float) * n * 3);
  hipMalloc(&x_last, sizeof(Float) * n * 3);
  hipMallocManaged(&v, sizeof(Float) * n * 3);
  hipMemset(v, 0, sizeof(Float) * n * 3);
  hipMalloc(&density, sizeof(Float) * n);
  thrust::fill(thrust::device, density, density + n, density_0);
  hipMalloc(&density_err, sizeof(Float) * n);
  hipMalloc(&pressure, sizeof(Float) * n);
  hipMalloc(&accel, sizeof(Float) * n * 3);
  hipMallocManaged(&neighbors, sizeof(int) * n * MAX_NEIGHBORS);
}

PCISPH::~PCISPH() {
  hipFree(x);
  hipFree(x_last);
  hipFree(v);
  hipFree(density);
  hipFree(density_err);
  hipFree(pressure);
  hipFree(accel);
  hipFree(neighbors);
}

__device__ Float viscosity_laplacian(Float r_norm) {
  Float k = 45.0f / (M_PI * pow(H, 6));
  r_norm = min(r_norm, H);
  return k * (H - r_norm);
}

__device__ __host__ Float cubic_kernel(const Float r_norm) {
  Float k = 8 / (M_PI * pow(H, 3));
  Float q = r_norm / H;
  if (q <= 1.0) {
    if (q <= 0.5) {
      return k * (6 * pow(q, 3) - 6 * pow(q, 2) + 1);
    } else {
      return k * 2 * pow(1 - q, 3);
    }
  }
  return 0;
}

__global__ void compute_non_pressure_force(int n, Float *x, Float *v,
                                           Float *density, Float *accel,
                                           int *neighbors) {
  int i = get_tid();
  if (i < n) {
    int *nei = neighbors + i * MAX_NEIGHBORS;
    int *end = nei + MAX_NEIGHBORS;
    Float *acc = accel + i * 3;
    acc[0] = 0, acc[1] = -9.8, acc[2] = 0;
    for (; nei != end; nei++) {
      int j = *nei;
      if (j == -1)
        break;

      Float *xi = x + i * 3, *xj = x + j * 3;
      Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
      Float r_norm = sqrtf(r[0] * r[0] + r[1] * r[1] + r[2] * r[2]);
      Float k = viscosity * (particle_mass / density[j]) *
                viscosity_laplacian(r_norm);

      Float *vi = v + i * 3, *vj = v + j * 3;
      Float vj_vi[3] = {vj[0] - vi[0], vj[1] - vi[1], vj[2] - vi[2]};
      acc[0] += k * vj_vi[0], acc[1] += k * vj_vi[1], acc[2] += k * vj_vi[2];
    }
  }
}

__global__ void advect(int n, Float *x, Float *x_last, Float *v, Float *accel) {
  int i = get_tid();
  if (i < n) {
    v[i] = v[i] + accel[i] * fixed_delta_time;
    x[i] = x_last[i] + v[i] * fixed_delta_time;
  }
}

__global__ void advect_pressure(int n, Float *x, Float *x_last, Float *v,
                                Float *accel) {
  int i = get_tid();
  if (i < n) {
    v[i] = v[i] + accel[i] * fixed_delta_time;
    x[i] = x_last[i] + accel[i] * fixed_delta_time * fixed_delta_time;
  }
}

__global__ void predict_x(int n, Float *x, Float *x_last, Float *accel) {
  int i = get_tid();
  if (i < n) {
    x[i] = x_last[i] + accel[i] * fixed_delta_time * fixed_delta_time;
  }
}

__global__ void compute_density(int n, Float *x, Float *density,
                                Float *density_err, int *neighbors) {
  int i = get_tid();
  if (i < n) {
    int *ptr = neighbors + i * MAX_NEIGHBORS;
    int *end = ptr + MAX_NEIGHBORS;

    Float density_i = cubic_kernel(0);
    for (; ptr != end; ptr++) {
      int j = *ptr;
      if (j == -1)
        break;

      Float *xi = x + i * 3, *xj = x + j * 3;
      Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
      Float r_norm = sqrt(r[0] * r[0] + r[1] * r[1] + r[2] * r[2]);
      density_i += cubic_kernel(r_norm);
    }
    density_i = density_i * particle_mass;

    Float density_err_i = density_i - density_0;
    if (density_err_i < 0)
      density_err_i = 0;

    density[i] = density_i;
    density_err[i] = density_err_i;
  }
}

Float compute_density_err_max(int n, Float *density_err) {
  Float max = 0;
  for (int i = 0; i < n; i++) {
    if (density_err[i] > max)
      max = density_err[i];
  }
  return max;
}

__global__ void compute_pressure(int n, Float delta, Float *density_err,
                                 Float *pressure) {
  int i = get_tid();
  if (i < n) {
    pressure[i] += delta * density_err[i];
  }
}

__global__ void compute_pressure_accel(int n, Float *x, Float *pressure,
                                       Float *density, Float *pressure_accel,
                                       int *neighbors) {
  int i = get_tid();
  if (i < n) {
    int *ptr = neighbors + i * MAX_NEIGHBORS;
    int *end = ptr + MAX_NEIGHBORS;

    Float acc[3] = {0, 0, 0};
    for (; ptr != end; ptr++) {
      int j = *ptr;
      if (j == -1)
        break;

      Float *xi = x + i * 3, *xj = x + j * 3;
      Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
      Float r_norm = sqrt(r[0] * r[0] + r[1] * r[1] + r[2] * r[2]);

      if (r_norm > 1e-5 && r_norm <= H) {
        Float k = -density_0 * particle_mass *
                  (pressure[i] / pow(density[i], 2) +
                   pressure[j] / pow(density[j], 2)) *
                  (-45.0f / (M_PI * pow(H, 6))) * (H - r_norm) * (H - r_norm) /
                  r_norm;
        acc[0] += k * r[0], acc[1] += k * r[1], acc[2] += k * r[2];
      }
    }
    Float *acc_i = pressure_accel + i * 3;
    acc_i[0] = acc[0], acc_i[1] = acc[1], acc_i[2] = acc[2];
  }
}

__global__ void enforceBoundaryComponent(int n, Float *x, Float *v, Float xmin,
                                         Float xmax) {
  int i = get_tid();
  if (i < n) {
    i = i * 3;
    if (x[i] < xmin) {
      x[i] = xmin;
      v[i] *= -.3f;
    }
    if (x[i] > xmax) {
      x[i] = xmax;
      v[i] *= -.3f;
    }
  }
}

void PCISPH::solver() {
  buildGrid(n, x, xmin, ymin, zmin, grid_size_x, grid_size_y, grid_size_z,
            neighbors);
  compute_non_pressure_force<<<grid_every_particle, bs>>>(n, x, v, density,
                                                          accel, neighbors);

  advect<<<grid_every_component, bs>>>(3 * n, x_last, x, v, accel);
  hipMemset(pressure, 0, sizeof(Float) * n);
  hipMemset(accel, 0, sizeof(Float) * n * 3);

  Float density_err_max = density_0;
  int i = 0;
  for (; i < MAX_PRESSURE_ITERATIONS && density_err_max / density_0 > 0.01;
       i++) {
    predict_x<<<grid_every_component, bs>>>(3 * n, x, x_last, accel);
    compute_density<<<grid_every_particle, bs>>>(n, x, density, density_err,
                                                 neighbors);
    compute_pressure<<<grid_every_particle, bs>>>(n, delta, density_err,
                                                  pressure);
    compute_pressure_accel<<<grid_every_particle, bs>>>(n, x, pressure, density,
                                                        accel, neighbors);
    density_err_max =
        thrust::reduce(thrust::device, density_err, density_err + n, 0.0f,
                       thrust::maximum<Float>());
  }

  advect_pressure<<<grid_every_component, bs>>>(3 * n, x, x_last, v, accel);
  enforceBoundaryComponent<<<grid_every_particle, bs>>>(n, x, v, xmin, xmax);
  enforceBoundaryComponent<<<grid_every_particle, bs>>>(n, x + 1, v + 1, ymin,
                                                        ymax);
  enforceBoundaryComponent<<<grid_every_particle, bs>>>(n, x + 2, v + 2, zmin,
                                                        zmax);
  hipDeviceSynchronize();
}
