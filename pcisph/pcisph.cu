#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "pcisph.h"
#include <algorithm>
#include <cassert>
#include <cmath>
#include <cstring>
#include <iostream>
#include <thrust/extrema.h>
#include <vector>

#define get_tid() (blockDim.x * blockIdx.x + threadIdx.x)

__device__ __constant__ unsigned int n;
__device__ __constant__ Float xmin, xmax, ymin, ymax, zmin, zmax;
__device__ __constant__ unsigned int grid_size_x, grid_size_y, grid_size_z;
__device__ __constant__ unsigned int grid_size;
__device__ __constant__ Float delta;

PCISPH::PCISPH(const Float xmin, const Float xmax, const Float ymin,
               const Float ymax, const Float zmin, const Float zmax,
               const size_t grid_size_x, const size_t grid_size_y,
               const size_t grid_size_z)
    : xmin(xmin), xmax(xmax), ymin(ymin), ymax(ymax), zmin(zmin), zmax(zmax),
      grid_size_x(grid_size_x), grid_size_y(grid_size_y),
      grid_size_z(grid_size_z) {}

int bs = 256;
dim3 grid_every_particle;
dim3 grid_every_component;

dim3 grid_every_grid;

void PCISPH::init(int n, Float delta) {
  auto initRes = hipInit(0);
  if (initRes != hipSuccess) {
    std::cerr << "CUDA init failed" << std::endl;
    exit(1);
  }
  std::cerr << "CUDA init success" << std::endl;
  std::cerr << "For each particle: ";
  std::cerr << "bs = " << bs << ' ';
  grid_every_particle = (n + bs - 1) / bs;
  grid_every_component = (n * 3 + bs - 1) / bs;
  std::cerr << "grid = (" << grid_every_particle.x << ','
            << grid_every_particle.y << ',' << grid_every_particle.z << ')'
            << std::endl;
  std::cerr << "number of particles = " << n << std::endl;
  unsigned int grid_size = grid_size_x * grid_size_y * grid_size_z;
  std::cerr << "number of grid = " << grid_size << std::endl;
  grid_every_grid = (grid_size + bs - 1) / bs;

  this->n = n;
  this->delta = delta;
  hipMemcpyToSymbol(HIP_SYMBOL(::n), &n, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(::xmin), &xmin, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(::xmax), &xmax, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(::ymin), &ymin, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(::ymax), &ymax, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(::zmin), &zmin, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(::zmax), &zmax, sizeof(Float));
  hipMemcpyToSymbol(HIP_SYMBOL(::grid_size_x), &grid_size_x, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(::grid_size_y), &grid_size_y, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(::grid_size_z), &grid_size_z, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(::grid_size), &grid_size, sizeof(unsigned int));
  hipMemcpyToSymbol(HIP_SYMBOL(::delta), &delta, sizeof(Float));

  hipMallocManaged(&x, sizeof(Float) * n * 3);
  hipMalloc(&x_last, sizeof(Float) * n * 3);
  hipMalloc(&v, sizeof(Float) * n * 3);
  hipMemset(v, 0, sizeof(Float) * n * 3);
  hipMalloc(&density, sizeof(Float) * n);
  thrust::fill(thrust::device, density, density + n, density_0);
  hipMalloc(&density_err, sizeof(Float) * n);
  hipMalloc(&pressure, sizeof(Float) * n);
  hipMalloc(&accel, sizeof(Float) * n * 3);
  hipMalloc(&neighbors, sizeof(int) * n * MAX_NEIGHBORS);
  hipMallocManaged(&grid,
                    sizeof(int) * grid_size * (MAX_PARTICLE_IN_GRID + 1));
  hipMallocManaged(&hash, sizeof(unsigned int) * n);
}

PCISPH::~PCISPH() {
  hipFree(x);
  hipFree(x_last);
  hipFree(v);
  hipFree(density);
  hipFree(density_err);
  hipFree(pressure);
  hipFree(accel);
  hipFree(neighbors);
  hipFree(grid);
  hipFree(hash);
}

__device__ int clamp(int x, int a, int b) { return max(a, min(b, x)); }

__device__ __forceinline__ float sqr(float x) { return x * x; }

__device__ int3 coordToGridIndex(float x, float y, float z) {
  int3 index;
  index.x = clamp((int)((x - xmin) / H), 0, grid_size_x - 1);
  index.y = clamp((int)((y - ymin) / H), 0, grid_size_y - 1);
  index.z = clamp((int)((z - zmin) / H), 0, grid_size_z - 1);
  return index;
}

__device__ int gridIndexToHash(int3 index) {
  return index.x * grid_size_y * grid_size_z + index.y * grid_size_z + index.z;
}

__global__ void clearGrid(int grid[][MAX_PARTICLE_IN_GRID + 1]) {
  int i = get_tid();
  if (i < grid_size) {
    grid[i][0] = 0;
  }
}

__global__ void compute_hash(Float *x, unsigned int *hash) {
  int i = get_tid();
  if (i < n) {
    Float *xi = x + i * 3;
    int3 index = coordToGridIndex(xi[0], xi[1], xi[2]);
    hash[i] = gridIndexToHash(index);
  }
}

__global__ void compute_neighbor(int grid[][MAX_PARTICLE_IN_GRID + 1], Float *x,
                                 int *neighbors) {
  int pi = get_tid();
  if (pi < n) {
    int cnt = 0;
    Float *xi = x + pi * 3;
    int *nei = neighbors + pi * MAX_NEIGHBORS;
    int3 index = coordToGridIndex(xi[0], xi[1], xi[2]);
    for (int i = -1; i <= 1; i++) {
      for (int j = -1; j <= 1; j++) {
        for (int k = -1; k <= 1; k++) {
          int3 neighbor_index{index.x + i, index.y + j, index.z + k};
          if (neighbor_index.x < 0 || neighbor_index.x >= grid_size_x ||
              neighbor_index.y < 0 || neighbor_index.y >= grid_size_y ||
              neighbor_index.z < 0 || neighbor_index.z >= grid_size_z)
            continue;
          long long hash = gridIndexToHash(neighbor_index);
          for (int idx = 1; idx <= grid[hash][0]; idx++) {
            int pj = grid[hash][idx];

            if (pi == pj)
              continue;

            Float *xj = x + pj * 3;
            Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
            Float r_norm_sqr = r[0] * r[0] + r[1] * r[1] + r[2] * r[2];

            if (r_norm_sqr > H2)
              continue;
            nei[cnt++] = pj;
            if (cnt == MAX_NEIGHBORS)
              goto end;
          }
        }
      }
    }
  end:
    for (; cnt < MAX_NEIGHBORS; cnt++)
      nei[cnt] = -1;
  }
}

__device__ Float viscosity_laplacian(Float r_norm) {
  Float k = 45.0f / (M_PI * H6);
  r_norm = min(r_norm, H);
  return k * (H - r_norm);
}

const Float cubic_kernel0 = 8 / (M_PI * H3);
__device__ __host__ Float cubic_kernel(const Float r_norm) {
  Float k = 8 / (M_PI * H3);
  Float q = r_norm / H;
  if (q <= 1.0) {
    if (q <= 0.5) {
      return k * (6 * pow(q, 3) - 6 * pow(q, 2) + 1);
    } else {
      return k * 2 * pow(1 - q, 3);
    }
  }
  return 0;
}

__global__ void compute_non_pressure_force(Float *x, Float *v, Float *density,
                                           Float *accel, int *neighbors) {
  int i = get_tid();
  if (i < n) {
    int *nei = neighbors + i * MAX_NEIGHBORS;
    int *end = nei + MAX_NEIGHBORS;
    Float *acc = accel + i * 3;
    acc[0] = 0, acc[1] = -9.8, acc[2] = 0;
    for (; nei != end; nei++) {
      int j = *nei;
      if (j == -1)
        break;

      Float *xi = x + i * 3, *xj = x + j * 3;
      Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
      Float r_norm = norm3df(r[0], r[1], r[2]);
      Float k = viscosity * (particle_mass / density[j]) *
                viscosity_laplacian(r_norm);

      Float *vi = v + i * 3, *vj = v + j * 3;
      Float vj_vi[3] = {vj[0] - vi[0], vj[1] - vi[1], vj[2] - vi[2]};
      acc[0] += k * vj_vi[0], acc[1] += k * vj_vi[1], acc[2] += k * vj_vi[2];
    }
  }
}

__global__ void advect(int n, Float *x, Float *x_last, Float *v, Float *accel) {
  int i = get_tid();
  if (i < n) {
    v[i] = v[i] + accel[i] * fixed_delta_time;
    x[i] = x_last[i] + v[i] * fixed_delta_time;
  }
}

__global__ void advect_pressure(int n, Float *x, Float *x_last, Float *v,
                                Float *accel) {
  int i = get_tid();
  if (i < n) {
    v[i] = v[i] + accel[i] * fixed_delta_time;
    x[i] = x_last[i] + accel[i] * fixed_delta_time * fixed_delta_time;
  }
}

__global__ void predict_x(int n, Float *x, Float *x_last, Float *accel) {
  int i = get_tid();
  if (i < n) {
    x[i] = x_last[i] + accel[i] * (fixed_delta_time * fixed_delta_time);
  }
}

__global__ void compute_density(Float *x, Float *density, Float *density_err,
                                int *neighbors) {
  int i = get_tid();
  if (i < n) {
    int *ptr = neighbors + i * MAX_NEIGHBORS;
    int *end = ptr + MAX_NEIGHBORS;

    Float density_i = cubic_kernel0;
    for (; ptr != end; ptr++) {
      int j = *ptr;
      if (j == -1)
        break;

      Float *xi = x + i * 3, *xj = x + j * 3;
      Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
      Float r_norm = norm3df(r[0], r[1], r[2]);
      density_i += cubic_kernel(r_norm);
    }
    density_i = density_i * particle_mass;

    Float density_err_i = density_i - density_0;
    if (density_err_i < 0)
      density_err_i = 0;

    density[i] = density_i;
    density_err[i] = density_err_i;
  }
}

Float compute_density_err_max(int n, Float *density_err) {
  Float max = 0;
  for (int i = 0; i < n; i++) {
    if (density_err[i] > max)
      max = density_err[i];
  }
  return max;
}

__global__ void compute_pressure(Float *density_err, Float *pressure) {
  int i = get_tid();
  if (i < n) {
    pressure[i] += delta * density_err[i];
  }
}

__global__ void compute_pressure_accel(Float *x, Float *pressure,
                                       Float *density, Float *pressure_accel,
                                       int *neighbors) {
  int i = get_tid();
  if (i < n) {
    int *ptr = neighbors + i * MAX_NEIGHBORS;
    int *end = ptr + MAX_NEIGHBORS;

    Float acc[3] = {0, 0, 0};
    for (; ptr != end; ptr++) {
      int j = *ptr;
      if (j == -1)
        break;

      Float *xi = x + i * 3, *xj = x + j * 3;
      Float r[3] = {xi[0] - xj[0], xi[1] - xj[1], xi[2] - xj[2]};
      Float r_norm = norm3df(r[0], r[1], r[2]);

      if (r_norm > 1e-5 && r_norm <= H) {
        Float k =
            (density_0 * particle_mass * (45.0f / (M_PI * H6))) *
            (pressure[i] / sqr(density[i]) + pressure[j] / sqr(density[j])) *
            sqr(H - r_norm) / r_norm;
        acc[0] += k * r[0], acc[1] += k * r[1], acc[2] += k * r[2];
      }
    }
    Float *acc_i = pressure_accel + i * 3;
    acc_i[0] = acc[0], acc_i[1] = acc[1], acc_i[2] = acc[2];
  }
}

__global__ void enforceBoundaryComponent(int n, Float *x, Float *v, Float xmin,
                                         Float xmax) {
  int i = get_tid();
  if (i < n) {
    i = i * 3;
    if (x[i] < xmin) {
      x[i] = xmin;
      v[i] *= -.3f;
    }
    if (x[i] > xmax) {
      x[i] = xmax;
      v[i] *= -.3f;
    }
  }
}

void PCISPH::solver() {
  clearGrid<<<grid_every_grid, bs>>>(grid);
  compute_hash<<<grid_every_particle, bs>>>(x, hash);
  hipDeviceSynchronize();
  for (int i = 0; i < n; i++) {
    int hashi = hash[i];
    if (grid[hashi][0] < MAX_PARTICLE_IN_GRID)
      grid[hashi][++grid[hashi][0]] = i;
  }
  compute_neighbor<<<grid_every_particle, bs>>>(grid, x, neighbors);

  compute_non_pressure_force<<<grid_every_particle, bs>>>(x, v, density, accel,
                                                          neighbors);

  advect<<<grid_every_component, bs>>>(3 * n, x_last, x, v, accel);
  hipMemset(pressure, 0, sizeof(Float) * n);
  hipMemset(accel, 0, sizeof(Float) * n * 3);

  Float density_err_max = density_0;
  int i = 0;
  for (; i < MAX_PRESSURE_ITERATIONS && density_err_max / density_0 > 0.01;
       i++) {
    predict_x<<<grid_every_component, bs>>>(3 * n, x, x_last, accel);
    compute_density<<<grid_every_particle, bs>>>(x, density, density_err,
                                                 neighbors);
    compute_pressure<<<grid_every_particle, bs>>>(density_err, pressure);
    compute_pressure_accel<<<grid_every_particle, bs>>>(x, pressure, density,
                                                        accel, neighbors);
    density_err_max =
        thrust::reduce(thrust::device, density_err, density_err + n, 0.0f,
                       thrust::maximum<Float>());
  }

  advect_pressure<<<grid_every_component, bs>>>(3 * n, x, x_last, v, accel);
  enforceBoundaryComponent<<<grid_every_particle, bs>>>(n, x, v, xmin, xmax);
  enforceBoundaryComponent<<<grid_every_particle, bs>>>(n, x + 1, v + 1, ymin,
                                                        ymax);
  enforceBoundaryComponent<<<grid_every_particle, bs>>>(n, x + 2, v + 2, zmin,
                                                        zmax);
  hipDeviceSynchronize();
}
